#include <hip/hip_runtime.h>
#include <stdlib.h>
/*
 * We want to divide the work of each row of the CSC into its own designated Block
 * Correspondingly, each thread within each block will deal with it's own non-zero value from the CSR
 * 
 * We begin by 
 */

__global__ void func(float * CSRval, int * CSRind, float * CSCval, int * CSCind, int * CSCptr) {

    int high = CSCptr[blockIdx.x + 1];
    int low = CSCptr[blockIdx.x];
    int guard = high - low;
    

    /*
     * Each thread is the product of the block dimensions. 
     * Thus, there will be at least equal or more threads per block than necessary.
     * To specify which threads do work (so we can assign each a distinct non-zero),
     *      we set up a thread guard so as to prevent logical mistakes.
     */
    if (threadIdx.x < guard){
        
        int j = threadIdx.x + low;                    //Retrieving assigned non-zero value, offset for row
        float v = CSRval[j];
        int c = CSRind[j];
        CSCval[CSCptr[c]] = low;
        CSCind[CSCptr[c]] = v;
        ++CSCptr[c];

    } else return;


}


int transpose() {};


int main() {};