#include <hip/hip_runtime.h>
#include <stdlib.h>
/*
 * We want to divide the work of each row of the CSC into its own designated Block
 * Correspondingly, each thread within each block will deal with it's own non-zero value from the CSR
 * 
 * We begin by establishing j value (unique non-zero identifier),
 *      then doing a lookup into said value and index of CSR. 
 * We then set the transposed position of value and index (using ptr row value) to said value and index.
 * 
 * A Final increment of the ptr value in question is necessary, to point to the next spot in the chain,
 *      for the next value index pair to be slotted in. Changes to ptr non-binding, b/c we won't cpy it back.
 */

__global__ void func(float * CSRval, int * CSRind, float * CSCval, int * CSCind, int * CSCptr) {

    int high = CSCptr[blockIdx.x + 1];
    int low = CSCptr[blockIdx.x];
    int guard = high - low;

    /*
     * Each thread is the product of the block dimensions. 
     * Thus, there will be at least equal or more threads per block than necessary.
     * To specify which threads do work (so we can assign each a distinct non-zero),
     *      we set up a thread guard so as to prevent logical mistakes.
     */
    if (threadIdx.x < guard){
        
        int j = threadIdx.x + low;                      //Retrieving assigned non-zero value, offset for row
        float v = CSRval[j];                            //Grab future value
        int c = CSRind[j];                              //Grab future row
        /*
         * Set future value (position determined by current spot open in row) properly
         * Set future index similarly, with Block ID (row num in CSR) accordingly
         * Only works b/c we increment the open position within each row
         */
        CSCval[CSCptr[c]] = v;
        CSCind[CSCptr[c]] = low;
        ++CSCptr[c];

    } else return;
}

__global__ void func2(int * CSRind, int * CSCptr, int CSCrows, int nonzeros) {
    int k = threadIdx.x;
    if (k < CSCrows + 1) {
        CSCptr[k] = 0;
    }

    __syncthreads();

    int j = threadIdx.x;
    if (j < nonzeros) { 
        CSCptr[CSRind[j] + 1]++;
    }
}

int transpose() {};


int main() {};