#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <stdexcept>
#include <assert.h>

inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}
#define gpuErrorCheck(ans) { gpuAssert((ans), __FILE__, __LINE__); }

/*
 * We want to divide the work of each row of the CSC into its own designated Block
 * Correspondingly, each thread within each block will deal with it's own non-zero value from the CSR
 * 
 * We begin by establishing j value (unique non-zero identifier),
 *      then doing a lookup into said value and index of CSR. 
 * We then set the transposed position of value and index (using ptr row value) to said value and index.
 * 
 * A Final increment of the ptr value in question is necessary, to point to the next spot in the chain,
 *      for the next value index pair to be slotted in. Changes to ptr non-binding, b/c we won't cpy it back.
 */

typedef struct CSR {
    int nrows; // number of rows
    int ncols; // number of rows
    int * ind; // column ids
    float * val; // values
    int * ptr; // pointers (start of row in ind/val)\

    CSR()
    {
        nrows = ncols = 0;
        ind = nullptr;
        val = nullptr;
        ptr = nullptr;
    }

    void reserve(const int nrows, const int nnz)
    {
        if(nrows > this->nrows){
            if(ptr){
                ptr = (int*) realloc(ptr, sizeof(int) * (nrows+1));
            } else {
                ptr = (int*) malloc(sizeof(int) * (nrows+1));
                ptr[0] = 0;
            }
            if(!ptr){
                throw std::runtime_error("Could not allocate ptr array.");
            }
        }
        if(nnz > ptr[this->nrows]){
            if(ind){
                ind = (int*) realloc(ind, sizeof(int) * nnz);
            } else {
                ind = (int*) malloc(sizeof(int) * nnz);
            }
            if(!ind){
                throw std::runtime_error("Could not allocate ind array.");
            }
            if(val){
                val = (float*) realloc(val, sizeof(float) * nnz);
            } else {
                val = (float*) malloc(sizeof(float) * nnz);
            }
            if(!val){
                throw std::runtime_error("Could not allocate val array.");
            }
        }
        this->nrows = nrows;
    }

    ~CSR() {
        if (ind) {
            free(ind);
        }
        if (val) {
            free(val);
        }
        if (ptr) {
            free(ptr);
        }
    }
} CSR;

__global__ void transposition(float * CSRval, int * CSRind, float * CSCval, int * CSCind, int * CSCptr) {

    int high = CSCptr[blockIdx.x + 1];
    int low = CSCptr[blockIdx.x];
    int guard = high - low;

    /*
     * Each thread is the product of the block dimensions. 
     * Thus, there will be at least equal or more threads per block than necessary.
     * To specify which threads do work (so we can assign each a distinct non-zero),
     *      we set up a thread guard so as to prevent logical mistakes.
     */
    if (threadIdx.x < guard){
        
        int j = threadIdx.x + low;                      //Retrieving assigned non-zero value, offset for row
        float v = CSRval[j];                            //Grab future value
        int c = CSRind[j];                              //Grab future row
        /*
         * Set future value (position determined by current spot open in row) properly
         * Set future index similarly, with Block ID (row num in CSR) accordingly
         * Only works b/c we increment the open position within each row
         */
        CSCval[CSCptr[c]] = v;
        CSCind[CSCptr[c]] = low;
        ++CSCptr[c];

    } else return;
}

__global__ void func2(int * CSRind, int * CSCptr, int CSCrows, int nonzeros) {
    int k = blockIdx.x * blockDim.x + threadIdx.x;
    if (k < CSCrows + 1) {
        CSCptr[k] = 0;
    }

    __syncthreads();

    int j = blockIdx.x * blockDim.x + threadIdx.x;
    if (j < nonzeros) { 
        CSCptr[CSRind[j] + 1]++;
    }
}

/*
 * Transpose takes in a csr_t data structure and creates a new one, populating it with values representing the transposed matrix.
 * mat's data is copied over to the GPU. The Transposition occurs, and data is copied back into transposed.
 * 
 */

CSR * transpose(CSR * mat) {
    CSR * tranposed = new CSR();

    float * d_values;
    int * d_indices;
    int * d_ptr;
    float * dt_values;
    int * dt_indices;
    int * dt_ptr;
    gpuErrorCheck(hipMalloc(&d_values, sizeof(float) * size))
    gpuErrorCheck(hipMalloc(&d_indices, sizeof(int) * size))
    gpuErrorCheck(hipMalloc(&d_ptr, sizeof(int) * size))

    gpuErrorCheck(hipMalloc(&dt_values, sizeof(float) * size))
    gpuErrorCheck(hipMalloc(&dt_indices, sizeof(int) * size))
    gpuErrorCheck(hipMalloc(&dt_ptr, sizeof(int) * size))
    
    gpuErrorCheck(hipMemcpy(d_values, mat->val, sizeof(float) * size, hipMemcpyHostToDevice))
    gpuErrorCheck(hipMemcpy(d_indices, mat->ind, sizeof(int) * size, hipMemcpyHostToDevice))
    gpuErrorCheck(hipMemcpy(d_ptr, mat->ptr, sizeof(int) * size, hipMemcpyHostToDevice))

    // func2<<<>>>

    gpuErrorCheck(hipMemcpy(transposed->val, dt_values, sizeof(float) * size, hipMemcpyDeviceToHost))
    gpuErrorCheck(hipMemcpy(transposed->ind, dt_indices, sizeof(int) * size, hipMemcpyDeviceToHost))
    gpuErrorCheck(hipMemcpy(transposed->ptr, dt_ptr, sizeof(int) * size, hipMemcpyDeviceToHost))





    gpuErrorCheck(hipFree(d_values))
    gpuErrorCheck(hipFree(d_indices))
    gpuErrorCheck(hipFree(d_ptr))
    gpuErrorCheck(hipFree(dt_values))
    gpuErrorCheck(hipFree(dt_indices))
    gpuErrorCheck(hipFree(dt_ptr))
    
    delete transposed;
};


int main() {

};